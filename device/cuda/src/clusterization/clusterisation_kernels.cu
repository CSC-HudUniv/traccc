#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "traccc/cuda/clusterization/clusterisation_kernels.cuh"

#include "traccc/clusterization/device/connect_components.hpp"
#include "traccc/clusterization/device/count_cluster_cells.hpp"
#include "traccc/clusterization/device/create_measurements.hpp"
#include "traccc/clusterization/device/find_clusters.hpp"
#include "traccc/clusterization/device/form_spacepoints.hpp"

namespace traccc::cuda {
namespace kernels {

__global__ void find_clusters(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {

    device::find_clusters(threadIdx.x + blockIdx.x * blockDim.x, cells_view,
                          sparse_ccl_indices_view, clusters_per_module_view);
}

__global__ void count_cluster_cells(
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view,
    vecmem::data::vector_view<unsigned int> cluster_sizes_view) {

    device::count_cluster_cells(
        threadIdx.x + blockIdx.x * blockDim.x, sparse_ccl_indices_view,
        cluster_prefix_sum_view, cells_prefix_sum_view, cluster_sizes_view);
}

__global__ void connect_components(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view,
    cluster_container_types::view clusters_view) {

    device::connect_components(threadIdx.x + blockIdx.x * blockDim.x,
                               cells_view, sparse_ccl_indices_view,
                               cluster_prefix_sum_view, cells_prefix_sum_view,
                               clusters_view);
}
__global__ void create_measurements(
    const cell_container_types::const_view cells_view,
    cluster_container_types::const_view clusters_view,
    measurement_container_types::view measurements_view) {

    device::create_measurements(threadIdx.x + blockIdx.x * blockDim.x,
                                clusters_view, cells_view, measurements_view);
}

__global__ void form_spacepoints(
    measurement_container_types::const_view measurements_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        measurements_prefix_sum_view,
    spacepoint_container_types::view spacepoints_view) {

    device::form_spacepoints(threadIdx.x + blockIdx.x * blockDim.x,
                             measurements_view, measurements_prefix_sum_view,
                             spacepoints_view);
}
}
}